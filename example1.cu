
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 32

//código device
__global__ void soma_vetor(int *a, int *b, int *c ){
	int indice = blockIdx.x;
	if(indice < N)
		c[indice] = a[indice] + b[indice];
}

//código host
int main(){
	int a[N],b[N],c[N];
	int* dev_a;
	int* dev_b;
	int* dev_c;

	int tam = N*sizeof(int);

	//inicializando as variaveis do host:
	for(int i=0; i < N; i++){
		a[i] = i;
		b[i]= i*2;
	}

	//alocando espaço para as variaveis da GPU:
	hipMalloc((void**)&dev_a,tam);
	hipMalloc((void**)&dev_b,tam);
	hipMalloc((void**)&dev_c,tam);

	//copiando as variaveis da CPU para a GPU:
	hipMemcpy(dev_a, &a, tam, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, &b, tam, hipMemcpyHostToDevice);

	//chamada da função da gpu (kernel):
	soma_vetor<<<N,1>>>(dev_a, dev_b, dev_c);

	//copiando o resultado da GPU para a CPU:
	hipMemcpy(&c, dev_c, tam, hipMemcpyDeviceToHost);

	//vizualizando o resultado:
	for(int i=0; i<N; i++)
		printf("%d ",c[i]);
	printf("\n\n");

	//liberando a memoria na GPU:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
