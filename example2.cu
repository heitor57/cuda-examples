
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 33
#define M 8 // Número de threads por bloco

//código device
// blockDim.x é a dimensão do bloco, ou seja,
// a quantidade de threads por bloco.
__global__ void soma_vetor(int *a, int *b, int *c ){
	int indice = blockIdx.x*blockDim.x + threadIdx.x;
	if(indice < N)
		c[indice] = a[indice] + b[indice];
}

//código host
int main(){
	int a[N],b[N],c[N];
	int* dev_a;
	int* dev_b;
	int* dev_c;

	int tam = N*sizeof(int);

	//inicializando as variaveis do host:
	for(int i=0; i < N; i++){
		a[i] = i;
		b[i]= i*2;
	}

	//alocando espaço para as variaveis da GPU:
	hipMalloc((void**)&dev_a,tam);
	hipMalloc((void**)&dev_b,tam);
	hipMalloc((void**)&dev_c,tam);

	//copiando as variaveis da CPU para a GPU:
	hipMemcpy(dev_a, &a, tam, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, &b, tam, hipMemcpyHostToDevice);
	//chamada da função da gpu (kernel):
	// Número de blocos é igual a dimensão do vetor
	// divida pela dimensão do bloco. N/M
	soma_vetor<<<(N+M-1)/M, M>>>(dev_a, dev_b, dev_c);

	//copiando o resultado da GPU para a CPU:
	hipMemcpy(&c, dev_c, tam, hipMemcpyDeviceToHost);

	//vizualizando o resultado:
	for(int i=0; i<N; i++)
		printf("%d ",c[i]);
	printf("\n\n");

	//liberando a memoria na GPU:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
