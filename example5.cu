
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 6
#define R 2
//código device
// blockDim.x é a dimensão do bloco, ou seja,
// a quantidade de threads por bloco.


__global__ void soma_numeros_adjacentes(int *a, int* res){
   __shared__ int temp[N];
	
	int ind = threadIdx.x;
	if(ind < N){
		int initial_pos = ind - R;
		if(initial_pos < 0){
			initial_pos = 0;
		}

		int final_pos = ind + R + 1;
		if(final_pos >= N){
			final_pos = N;
		}
		temp[ind] = 0;
		for(int i = initial_pos; i < final_pos; i++){
			temp[ind] += a[i];
		}
	}
	res[ind] = temp[ind];

}

//código host
int main(){
	int a[N];
	int res[N];
	int* dev_a;
	int* dev_res;

	int tam = N*sizeof(int);

	//inicializando as variaveis do host:
	for(int i=0; i < N; i++){
		a[i] = i+1;
		printf("%d ",a[i]);
	}
	printf("\n");
	//alocando espaço para as variaveis da GPU:
	hipMalloc((void**)&dev_a,tam);
	hipMalloc((void**)&dev_res,tam);

	//copiando as variaveis da CPU para a GPU:
	hipMemcpy(dev_a, &a, tam, hipMemcpyHostToDevice);
	hipMemcpy(dev_res, &res, tam, hipMemcpyHostToDevice);
	//chamada da função da gpu (kernel):
	// Número de blocos é igual a dimensão do vetor
	// divida pela dimensão do bloco. N/M

	// O tipo dim3 permite definir a quantidade de
	// blocos e threads por dimensão
	//dim3 numBlocos(2,2);// número de blocos é igual a 2x2 = 4
	//dim3 numThreads(1,2);// número de threads por bloco = 2x2 = 4

	dim3 numBlocos(1);
	dim3 numThreads(N);
	soma_numeros_adjacentes<<<numBlocos,numThreads>>>(dev_a, dev_res);

	//copiando o resultado da GPU para a CPU:
	hipMemcpy(&res, dev_res, tam, hipMemcpyDeviceToHost);

	//vizualizando o resultado:

	for(int i=0; i < N; i++){
		a[i] = i;
		printf("%d ",res[i]);
	}

	printf("\n\n");

	//liberando a memoria na GPU:
	hipFree(dev_a);
	hipFree(dev_res);

	return 0;
}
